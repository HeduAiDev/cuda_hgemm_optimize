#include "hip/hip_runtime.h"


#include <stddef.h>    // for size_t
#include <array>       // for array
#include <atomic>      // for atomic
#include <chrono>      // for operator""s, chrono_literals
#include <cmath>       // for sin
#include <functional>  // for ref, reference_wrapper, function
#include <memory>      // for allocator, shared_ptr, __shared_ptr_access
#include <string>  // for string, basic_string, char_traits, operator+, to_string
#include <thread>   // for sleep_for, thread
#include <utility>  // for move
#include <vector>   // for vector
#include <sstream>
 
#include "ftxui/component/component.hpp"  // for Checkbox, Renderer, Horizontal, Vertical, Input, Menu, Radiobox, ResizableSplitLeft, Tab
#include "ftxui/component/component_base.hpp"  // for ComponentBase, Component
#include "ftxui/component/component_options.hpp"  // for MenuOption, InputOption
#include "ftxui/component/event.hpp"              // for Event, Event::Custom
#include "ftxui/component/screen_interactive.hpp"  // for Component, ScreenInteractive
#include "ftxui/dom/elements.hpp"  // for text, color, operator|, bgcolor, filler, Element, vbox, size, hbox, separator, flex, window, graph, EQUAL, paragraph, WIDTH, hcenter, Elements, bold, vscroll_indicator, HEIGHT, flexbox, hflow, border, frame, flex_grow, gauge, paragraphAlignCenter, paragraphAlignJustify, paragraphAlignLeft, paragraphAlignRight, dim, spinner, LESS_THAN, center, yframe, GREATER_THAN
#include "ftxui/dom/flexbox_config.hpp"  // for FlexboxConfig
#include "ftxui/screen/color.hpp"  // for Color, Color::BlueLight, Color::RedLight, Color::Black, Color::Blue, Color::Cyan, Color::CyanLight, Color::GrayDark, Color::GrayLight, Color::Green, Color::GreenLight, Color::Magenta, Color::MagentaLight, Color::Red, Color::White, Color::Yellow, Color::YellowLight, Color::Default, Color::Palette256, ftxui
#include "ftxui/screen/color_info.hpp"  // for ColorInfo
#include "ftxui/screen/terminal.hpp"    // for Size, Dimensions
#include "tui_tool_sets.hpp"

using namespace ftxui;

int main()
{
    auto screen = ScreenInteractive::Fullscreen();
    /////////////////////////////////////////////////////////////////////
    //block1
    /////////////////////////////////////////////////////////////////////

    // kernel selector
    std::vector<std::string> kernel_available{};
    for(int i = 0; i < 20; i++) {
        kernel_available.push_back("kernel" + std::to_string(i));
    };
    int kernel_selected = 0;
    tui::component::RadioFrameOptions radioframe_options;
    radioframe_options.max_height = 5;
    radioframe_options.title_regx = "kernel:%s";
    Component kernel_selector = tui::component::RadioFrame(&kernel_available, &kernel_selected, radioframe_options) | border;

    // runtime info
    bool ready = false;
    std::string info = 
        "simple usage: "
        " First choose a kernel."
        " Second input configs."
        " Third click run button.";
    Component run_button = Button("run", [&] {
        if (!ready) {
            info = "Please input config first";
            return;
        }
    }, ButtonOption::Animated());

    Component info_block  = Renderer(run_button ,[&] {
        return hbox({
            paragraph(info),
            separator(),
            run_button -> Render()
        }) | borderRounded;
    });

    // config panel
    std::string config_m;
    std::string config_n;
    std::string config_k;
    std::string config_tile_m;
    std::string config_tile_n;
    std::string config_tile_k;
    std::string config_launch_cnt;

    auto input_transform = [](InputState state)
    {
        state.element |= borderRounded;
        state.element |= color(Color::White);

        if (state.is_placeholder)
        {
            state.element |= dim;
        }

        if (state.focused)
        {
            state.element |= bgcolor(Color::Black);
        }

        if (state.hovered)
        {
            state.element |= bgcolor(Color::GrayDark);
        }

        return state.element;
    };


    auto input_style = [](Element ele) { return ele | size(WIDTH, LESS_THAN, 12) | size(WIDTH, GREATER_THAN, 7) | size(HEIGHT, LESS_THAN, 5); };
    auto label_style = [](Element ele) { return ele | align_right | vcenter ;};

    auto input_cell = [&] (std::string label, ftxui::StringRef constent, std::string placeholder, tui::component::InputType inputType, std::function<Element(InputState)> transform)
    { 
        tui::component::InputElementConfig input_config;
        input_config.label = label;
        input_config.input_type = inputType;
        input_config.placeholder = placeholder;
        input_config.content = std::move(constent);
        input_config.transform = transform;
        input_config.input_style = input_style;
        input_config.label_style = label_style;
        return input_config;
    };
    

    Component input_form =  tui::component::InputForm({
        {
            input_cell("M :", &config_m, "M", tui::component::InputType::Number, input_transform),
            input_cell("N :", &config_n, "N", tui::component::InputType::Number, input_transform),
            input_cell("K :", &config_k, "K", tui::component::InputType::Number, input_transform),
        },
        {
            input_cell("TileM :", &config_tile_m, "TileM", tui::component::InputType::Number, input_transform),
            input_cell("TileN :", &config_tile_n, "TileN", tui::component::InputType::Number, input_transform),
            input_cell("TileK :", &config_tile_k, "TileK", tui::component::InputType::Number, input_transform),
        },
    });
    Component input_config_launch_cnt = Input(&config_launch_cnt, "launch count"); 
    
    input_form -> ChildAt(0) -> Add(input_config_launch_cnt);
    Component config_panel = Renderer(input_form, [&] {
        return window(text("config") | hcenter | bold, vbox({
            input_form -> Render(),
            hbox({label_style(text("Launch cnt :")), input_style(input_config_launch_cnt -> Render())})
        }))  | xflex_grow;
    });

    Component block1 = Renderer(
        Container::Vertical({
            kernel_selector,
            info_block,
            config_panel
        }),
        [&] {
        return vbox({
            hbox({kernel_selector -> Render(), info_block -> Render() | flex}),
            config_panel -> Render(),
        }) | flex;
    });

    /////////////////////////////////////////////////////////////////////
    //block2
    /////////////////////////////////////////////////////////////////////

    // auto make_box = [](std::string val) {
    //     return text(val) | center | frame | size(WIDTH, EQUAL, 3) | size(HEIGHT, EQUAL, 1) ;
    // };
    // Box b_box;
    // auto make_grid = [&](int* ptr, int rows, int cols) {
    //     std::vector<Elements> crows;
    //     for (int i = 0; i < rows; i++) {
    //         std::vector<Element> ccols;
    //         std::vector<Element> row_separator;
    //         for (int j = 0; j < cols; j++) {
    //             Element e = make_box(std::to_string(ptr[i * cols + j]));
    //             // if (i == 0) {
    //             //     e = e | reflect(boxs[j]);
    //             // }
    //             ccols.push_back(e);
    //             ccols.push_back(separator());
    //             row_separator.push_back(separator());
    //             row_separator.push_back(separator());
    //         }
    //         crows.push_back(ccols);
    //         if (i != rows - 1) {
    //             crows.push_back(row_separator);
    //         }
            
    //     }
    //     return gridbox(crows) | reflect(b_box);
    // };

    // float focus_x = 0.5f;
    // float focus_y = 0.5f;
    // SliderOption<float> slider_x_option = {&focus_x, 0.0f, 1.0f, 0.01f, Direction::Right, Color::White, Color::Grey50};
    // SliderOption<float> slider_y_option = {&focus_y, 0.0f, 1.0f, 0.01f, Direction::Down, Color::White, Color::Grey50};
    // auto slider_x = Slider(slider_x_option) | bgcolor(Color::Grey23);
    // auto slider_y = Slider(slider_y_option) | bgcolor(Color::Grey23);

    // int* b_ptr = new int[128 * 128];

    // for (int row = 0; row < 128; row++) {
    //     for (int col = 0; col < 128; col++) {
    //         b_ptr[row * 128 + col] = row;
    //     }
    // }
    

    // Component block2 = Container::Vertical({
    //     slider_x,
    //     slider_y
    // });

    // Element matrix_b = make_grid(b_ptr, 128, 128);


    // std::vector<Element> col_labels;
    // for (int i = 0; i < 128; i++) {
    //     col_labels.push_back(text(std::to_string(i)) | center | frame | size(WIDTH, EQUAL, 3) | color(Color::Gold3Bis) | bgcolor(Color::Grey3));
    //     col_labels.push_back(separator() | color(Color::Gold3) | bgcolor(Color::Grey3));
    //     // col_labels.push_back(text("  ") | bgcolor(Color::Grey3));
    // }
    // Element matrix_b_col_labels = gridbox({col_labels});

    // std::vector<std::vector<Element>> row_labels;
    // for (int i = 0; i < 127; i++) {
    //     row_labels.push_back({text(std::to_string(i)) | size(HEIGHT, EQUAL, 1) | center | color(Color::Gold3Bis) | bgcolor(Color::Grey3)});
    //     row_labels.push_back({separator() | color(Color::Gold3) | bgcolor(Color::Grey3)});
    // }
    // row_labels.push_back({text(std::to_string(127)) | size(HEIGHT, EQUAL, 1) | center | color(Color::Gold3Bis) | bgcolor(Color::Grey3)});

    // Element matrix_b_row_labels = gridbox(row_labels);

    // Component block2_renderer = Renderer(block2,
    //                                      [&]
    //                                      {
    //                                          return window(
    //                                              text("matrixB") | hcenter | bold,

    //                                             vbox({
    //                                                 hbox({
    //                                                     vbox({
    //                                                         slider_x->Render() | size(HEIGHT, EQUAL, 1),
    //                                                         gridbox({
    //                                                             {matrix_b_col_labels | focusPositionRelative(focus_x, 0) | frame | size(HEIGHT, EQUAL, 1)},
    //                                                             {
    //                                                                 matrix_b | focusPositionRelative(focus_x, focus_y) | frame,
    //                                                             },
    //                                                         }),
    //                                                     }) | flex,
    //                                                     vbox({
    //                                                         text(" ") | size(HEIGHT, EQUAL, 2),
    //                                                         hbox({
    //                                                             matrix_b_row_labels | focusPositionRelative(0, focus_y) | frame,
    //                                                             slider_y->Render()
    //                                                         }) | yflex 
    //                                                     }) | size(WIDTH, EQUAL, 4)

    //                                                 })
    //                                             })
    //                                          );
    //                                      });

    int row = 1024;
    int col = 1024;
    float* matrix_b_ptr = new float[row * col];
    Component block2 = ::tui::component::MatrixFrame(matrix_b_ptr, row, col);



    Component block3 = Renderer([] { return text("block3") | center | flex;});
    Component block4 = Renderer([] { return text("block4") | center | flex;});



    tui::component::Resizable4BlockOptions options;
    // options.placeholder_block1 = text("Redraw matrix is inefficient") | center | bold;
    options.placeholder_block2 = text("Redraw matrix is inefficient") | center | bold;

    Component menu1_renderer = tui::component::Resizable4Block(block1, block2, block3, block4, screen, options);

    int tab_index = 0;
    std::vector<std::string> tab_entries = {
        "menu1", "menu2", "menu3"
    };
    auto tab_section = Menu(&tab_entries, &tab_index, MenuOption::HorizontalAnimated());
    auto tab_content = Container::Tab({
        menu1_renderer,
        // block2,
        // block3
    }, &tab_index);
    Component main_container = Container::Vertical({
        tab_section,
        tab_content
    });
    Component main_renderer = Renderer(main_container, [&] {
        return vbox({
            text("Demo" + config_tile_m +":"+ config_m +":"+ config_launch_cnt) | bold | hcenter,
            tab_section -> Render(),
            tab_content -> Render()
        });
    });
    screen.Loop(main_renderer);
    return 0;
}