#include "hip/hip_runtime.h"


#include <stddef.h>    // for size_t
#include <array>       // for array
#include <atomic>      // for atomic
#include <chrono>      // for operator""s, chrono_literals
#include <cmath>       // for sin
#include <functional>  // for ref, reference_wrapper, function
#include <memory>      // for allocator, shared_ptr, __shared_ptr_access
#include <string>  // for string, basic_string, char_traits, operator+, to_string
#include <thread>   // for sleep_for, thread
#include <utility>  // for move
#include <vector>   // for vector
#include <sstream>
 
#include "ftxui/component/component.hpp"  // for Checkbox, Renderer, Horizontal, Vertical, Input, Menu, Radiobox, ResizableSplitLeft, Tab
#include "ftxui/component/component_base.hpp"  // for ComponentBase, Component
#include "ftxui/component/component_options.hpp"  // for MenuOption, InputOption
#include "ftxui/component/event.hpp"              // for Event, Event::Custom
#include "ftxui/component/screen_interactive.hpp"  // for Component, ScreenInteractive
#include "ftxui/dom/elements.hpp"  // for text, color, operator|, bgcolor, filler, Element, vbox, size, hbox, separator, flex, window, graph, EQUAL, paragraph, WIDTH, hcenter, Elements, bold, vscroll_indicator, HEIGHT, flexbox, hflow, border, frame, flex_grow, gauge, paragraphAlignCenter, paragraphAlignJustify, paragraphAlignLeft, paragraphAlignRight, dim, spinner, LESS_THAN, center, yframe, GREATER_THAN
#include "ftxui/dom/flexbox_config.hpp"  // for FlexboxConfig
#include "ftxui/screen/color.hpp"  // for Color, Color::BlueLight, Color::RedLight, Color::Black, Color::Blue, Color::Cyan, Color::CyanLight, Color::GrayDark, Color::GrayLight, Color::Green, Color::GreenLight, Color::Magenta, Color::MagentaLight, Color::Red, Color::White, Color::Yellow, Color::YellowLight, Color::Default, Color::Palette256, ftxui
#include "ftxui/screen/color_info.hpp"  // for ColorInfo
#include "ftxui/screen/terminal.hpp"    // for Size, Dimensions
#include "tui_tool_sets.hpp"

using namespace ftxui;

int main()
{
    auto screen = ScreenInteractive::Fullscreen();
    /////////////////////////////////////////////////////////////////////
    //block1
    /////////////////////////////////////////////////////////////////////
    // kernel selector
    std::vector<std::string> kernel_available{};
    for(int i = 0; i < 20; i++) {
        kernel_available.push_back("kernel" + std::to_string(i));
    };
    int kernel_selected = 0;
    Component kernel_selector1 = Radiobox(&kernel_available, &kernel_selected) | vscroll_indicator | frame | size(HEIGHT, LESS_THAN, 5);
    Component kernel_selector = Renderer(kernel_selector1, [&] {
        return vbox({
            hbox({text("kernel:"), text(kernel_available[kernel_selected]) | bold }),
            separator(),
            kernel_selector1->Render()
        }) | border;
    });

    // runtime info
    bool ready = false;
    std::string info = 
        "simple usage: "
        " First choose a kernel."
        " Second input configs."
        " Third click run button.";
    Component run_button = Button("run", [&] {
        if (!ready) {
            info = "Please input config first";
            return;
        }
    }, ButtonOption::Animated());

    Component info_block  = Renderer(run_button ,[&] {
        return hbox({
            paragraph(info),
            separator(),
            run_button -> Render()
        }) | borderRounded;
    });

    // config panel
    std::string config_m;
    std::string config_n;
    std::string config_k;
    std::string config_tile_m;
    std::string config_tile_n;
    std::string config_tile_k;
    std::string config_launch_cnt;

    // InputOption input_style = InputOption::Spacious();
    InputOption input_option;
    input_option.transform = [](InputState state)
    {
        state.element |= borderRounded;
        state.element |= color(Color::White);

        if (state.is_placeholder)
        {
            state.element |= dim;
        }

        if (state.focused)
        {
            state.element |= bgcolor(Color::Black);
        }

        if (state.hovered)
        {
            state.element |= bgcolor(Color::GrayDark);
        }

        return state.element;
    };
    Component input_config_m = Input(&config_m, "M", input_option);
    Component input_config_n = Input(&config_n, "N", input_option);
    Component input_config_k = Input(&config_k, "K", input_option);
    Component input_config_tile_m = Input(&config_tile_m, "TileM", input_option);
    Component input_config_tile_n = Input(&config_tile_n, "TileN", input_option);
    Component input_config_tile_k = Input(&config_tile_k, "TileK", input_option);
    Component input_config_launch_cnt = Input(&config_launch_cnt, "launch count"); 
    
    input_config_m |= CatchEvent([&](Event event) {
        return event.is_character() && !std::isdigit(event.character()[0]);
    });
    input_config_n |= CatchEvent([&](Event event) {
        return event.is_character() && !std::isdigit(event.character()[0]);
    });
    input_config_k |= CatchEvent([&](Event event) {
        return event.is_character() && !std::isdigit(event.character()[0]);
    });
    input_config_tile_m |= CatchEvent([&](Event event) {
        return event.is_character() && !std::isdigit(event.character()[0]);
    });
    input_config_tile_n |= CatchEvent([&](Event event) {
        return event.is_character() && !std::isdigit(event.character()[0]);
    });
    input_config_tile_k |= CatchEvent([&](Event event) {
        return event.is_character() && !std::isdigit(event.character()[0]);
    });
    input_config_launch_cnt |= CatchEvent([&](Event event) {
        return event.is_character() && !std::isdigit(event.character()[0]);
    });
    Component config_panel = Container::Vertical({
        Container::Horizontal({
            input_config_m,
            input_config_n,
            input_config_k,
        }),
        Container::Horizontal({
            input_config_tile_m,
            input_config_tile_n,
            input_config_tile_k,
        }),
        input_config_launch_cnt
    });

    auto input = [](Element ele) { return ele | size(WIDTH, LESS_THAN, 12) | size(WIDTH, GREATER_THAN, 7) | size(HEIGHT, LESS_THAN, 5); };
    auto h = [](Element ele) { return ele | align_right | vcenter ;};
    Component config_panel_renderer = Renderer(config_panel, [&] {
        return window(text("config") | hcenter | bold, vbox({
            gridbox({
                {h(text(" M :")), input(input_config_m -> Render()), h(text(" N :")), input(input_config_n -> Render()), h(text(" K :")), input(input_config_k -> Render())},
                {h(text(" TileM :")), input(input_config_tile_m -> Render()), h(text(" TileN :")), input(input_config_tile_n -> Render()), h(text(" TileK :")), input(input_config_tile_k -> Render())},
            }),
            hbox({h(text("Launch cnt :")) , input(input_config_launch_cnt -> Render())})
        }))  | xflex_grow;

    });


    Component block1 = Container::Vertical({
        kernel_selector,
        info_block,
        config_panel_renderer
    });

    Component block1_renderer = Renderer(block1, [&] {
        return vbox({
            hbox({kernel_selector -> Render(), info_block -> Render() | flex}),
            config_panel_renderer -> Render(),
        }) | flex;
    });

    /////////////////////////////////////////////////////////////////////
    //block2
    /////////////////////////////////////////////////////////////////////

    auto make_box = [](std::string val) {
        return text(val) | center | size(WIDTH, EQUAL, 3) | size(HEIGHT, EQUAL, 1) | border;
    };

    auto make_grid = [&](int* ptr, int rows, int cols) {
        std::vector<Elements> crows;
        for (int i = 0; i < rows; i++) {
            std::vector<Element> ccols;
            for (int j = 0; j < cols; j++) {
                ccols.push_back(make_box(std::to_string(ptr[i * cols + j])));
            }
            crows.push_back(ccols);
        }
        return gridbox(crows);
    };

    float focus_x = 0.5f;
    float focus_y = 0.5f;
    SliderOption<float> slider_x_option = {&focus_x, 0.0f, 1.0f, 0.01f};
    SliderOption<float> slider_y_option = {&focus_y, 0.0f, 1.0f, 0.01f, Direction::Down};
    auto slider_x = Slider(slider_x_option);
    auto slider_y = Slider(slider_y_option);

    int* b_ptr = new int[128 * 128];

    Component block2 = Container::Vertical({
        slider_x,
        slider_y
    });

    Element matrix_b = make_grid(b_ptr, 128, 128);

    Component block2_renderer = Renderer(block2,
                                         [&]
                                         {
                                             return window(
                                                 text("matrixB") | hcenter | bold,
                                                 vbox({slider_x->Render() | size(HEIGHT, EQUAL, 1),
                                                       hbox({matrix_b | focusPositionRelative(focus_x, focus_y) | frame | flex,
                                                             slider_y->Render()})}));
                                         });
    Component block3 = Renderer([] { return text("block3") | center | flex;});
    Component block4 = Renderer([] { return text("block4") | center | flex;});



    tui::component::Resizable4BlockOptions options;
    // options.placeholder_block1 = text("Redraw matrix is inefficient") | center | bold;
    options.placeholder_block2 = text("Redraw matrix is inefficient") | center | bold;

    Component menu1_renderer = Resizable4Block(block1_renderer, block2_renderer, block3, block4, screen, options);

    int tab_index = 0;
    std::vector<std::string> tab_entries = {
        "menu1", "menu2", "menu3"
    };
    auto tab_section = Menu(&tab_entries, &tab_index, MenuOption::HorizontalAnimated());
    auto tab_content = Container::Tab({
        menu1_renderer,
        // block2,
        // block3
    }, &tab_index);
    Component main_container = Container::Vertical({
        tab_section,
        tab_content
    });
    Component main_renderer = Renderer(main_container, [&] {
        return vbox({
            text("Demo") | bold | hcenter,
            tab_section -> Render(),
            tab_content -> Render()
        });
    });
    screen.Loop(main_renderer);
    return 0;
}