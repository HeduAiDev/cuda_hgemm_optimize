#include <hipblas.h>
#include "gemm/base.hpp"
#include "utils/tensor.hpp"
#include "gemm.cuh"

gemm::base::GemmOutput cuBLASGemm(half *A_ptr, half *B_ptr, half *C_ptr, int M, int N, int K )
{
    using namespace utils::tensor;
    utils::Timeit t;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // malloc on device
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);
    
    Tensor<half> A = Tensor<half>( A_ptr, M, K, StorageOrder::RowMajor );
    Tensor<half> B = Tensor<half>( B_ptr, K, N, StorageOrder::RowMajor );
    Tensor<half> C = Tensor<half>( C_ptr, M, N, StorageOrder::RowMajor );
    
    A.copyToDevice();
    B.copyToDevice();
    C.copyToDevice();

    half alpha = static_cast<half>(1);
    half beta = static_cast<half>(0);

    t.start();
   hipblasStatus_t status = hipblasGemmEx(handle,
                                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                                 N, M, K,
                                 &alpha,
                                 B.devicePtr(), HIP_R_16F, N,
                                 A.devicePtr(), HIP_R_16F, K,
                                 &beta,
                                 C.devicePtr(), HIP_R_16F, N, HIPBLAS_COMPUTE_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    t.stop();
    gemm::base::GemmOutput options;
    options.status = gemm::base::Status::Success;
    options.excute_time_ms = t.elapsed_time;
    if (status != HIPBLAS_STATUS_SUCCESS)
    {
        options.status = gemm::base::Status::Error;
        options.code = (hipError_t)status;
        options.err = "CuBlas error: " + std::string(cublasGetStatusString(status)) + "\n" + __FILE__ + ":" + std::to_string(__LINE__) + "\n";
    }
    C.copyToHost();
    hipblasDestroy(handle);
    return options;
}