#include "hip/hip_runtime.h"
#include "gemm.cuh"
#include "utils/tensor.hpp"
using namespace gemm::base;

#define BlockTileM 16
#define BlockTileN 8


// 14.829097 ms, M=N=2048, K=1024
__global__ void simt_naive_kernel(half* __restrict__ A, half* __restrict__ B, half* __restrict__ C, int M, int N, int K) {
    int offset_st_global_cx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset_st_global_cy = blockIdx.y * blockDim.y + threadIdx.y;
    int offset_ld_reg1_a = offset_st_global_cy;
    int offset_ld_reg1_b = offset_st_global_cx;
    half sum = 0;
    for (int k = 0; k < K; k++) {
        half reg1_a = A[offset_ld_reg1_a * K + k];
        half reg1_b = B[k * N + offset_ld_reg1_b];
        sum += reg1_a * reg1_b;
    }
    C[offset_st_global_cy * N + offset_st_global_cx] = sum;
};




gemm::base::GemmOutput simt_naive(half* A_ptr, half *B_ptr, half *C_ptr, int M, int N, int K, const int launch_times) {
    using namespace utils::tensor;
    Tensor<half> A = Tensor<half>( A_ptr, M, K, StorageOrder::RowMajor );
    Tensor<half> B = Tensor<half>( B_ptr, K, N, StorageOrder::RowMajor );
    Tensor<half> C = Tensor<half>( C_ptr, M, N, StorageOrder::RowMajor );
    
    A.copyToDevice();
    B.copyToDevice();
    C.copyToDevice();

    dim3 grid(divCeil(N, BlockTileN), divCeil(M, BlockTileM));
    dim3 block(BlockTileN, BlockTileM);
    utils::Timeit t;
    for (int i = 0; i < launch_times; i++) {
        t.start();
        simt_naive_kernel<<<grid, block>>>(A.devicePtr(), B.devicePtr(), C.devicePtr(), M, N, K);
        // gmem_kernel<<<grid, block>>>(A.devicePtr(), B.devicePtr(), C.devicePtr(), M, N, K);
        t.stop();
        C.initializeHostData(InitializationType::Zero);
        C.copyToHost();
    }
    return gemm::base::GemmOutput(hipGetLastError(), t.cumulative_time / launch_times);
};
